#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define DATA_SIZE 10

__global__ void quicksort(int *data,int left,int right){
    unsigned int thread_id=threadIdx.x
    int pivot;
    int min,max;
    int i;
    for(i=0;i<DATA_SIZE;i++){
    }
}

__device__ void ave(int *data,int size){
    

}
 
__host__ void launch_quicksort(int *data,int count){
    quicksort<<<1,1>>>(data,0,count-1);
}

int main(int argc,char *argv[]){
    int list[DATA_SIZE];
    int *dev_c;
    int i;
    hipError_t err;
    
    srand(1);

    for(i=0;i<DATA_SIZE;i++)
        list[i]=rand()%20;

    err=hipMalloc((void **)&dev_c,sizeof(int)*DATA_SIZE);
    if(err != hipSuccess){
        printf("hipMalloc Error\n");
    }

    launch_quicksort<<<1,1>>>(dev_c);

    
    err=hipMemcpy(&dev_c,list,sizeof(int)*DATA_SIZE,hipMemcpyDeviceToHost);
    if(err != hipSuccess){
        printf("hipMalloc Error\n");
    }
    
    for(i=0;i<DATA_SIZE;i++)
        printf("%d\n",list[i]);

    
    return 0;
}
