
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <stdlib.h>

#define DATA_SIZE 10

__global__ void cusum(int *data,int *size,float *sum){
    int thi=threadIdx.x;

    if(thi<DATA_SIZE){
        sum+=data[thi];
    }
    
}

int main(int argc,char *argv[]){
    int *list;
    int *dev_list;
    int i;
    int size=DATA_SIZE;
    int *dev_size;
    float sum=0;
    float *dev_sum;
    hipError_t err;
    
    srand(1);
    
    list=(int *)malloc(sizeof(int)*DATA_SIZE);
    for(i=0;i<DATA_SIZE;i++){
        list[i]=rand()%20;
        printf("%d\n",list[i]);
    }

    err=hipMalloc((void **)&dev_list,sizeof(int)*DATA_SIZE);
    err=hipMalloc((void **)&dev_sum,sizeof(float));
    err=hipMalloc((void **)&dev_size,sizeof(int));

    err=hipMemcpy(dev_list,list,sizeof(int)*DATA_SIZE,hipMemcpyHostToDevice);
    err=hipMemcpy(dev_sum,&sum,sizeof(float),hipMemcpyHostToDevice);
    err=hipMemcpy(dev_size,&size,sizeof(float),hipMemcpyHostToDevice);
    
    if(err != hipSuccess){
        printf("hipMalloc Error\n");
    }

    cusum<<<1,10>>>(dev_list,dev_size,dev_sum);

    err=hipMemcpy(&sum,dev_sum,sizeof(float),hipMemcpyDeviceToHost);
    if(err != hipSuccess){
        printf("hipMalloc Error\n");
    }

    hipFree(dev_sum);
    hipFree(dev_list);
    hipFree(dev_size);
    free(list);
    printf("sum=%f\n",sum);
    
    return 0;
}
